#include "hip/hip_runtime.h"
#include "../GpuCurve.h"
#include "../GpuDelaunay.h"

#include <iomanip>
#include <iostream>

#include "KerCommon.h"
#include "KerDivision.h"
#include "KerPredicates.h"
#include "ThrustWrapper.h"

#define GRID_STRIDE_LOOP(i, n)                                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                 \
       i += blockDim.x * gridDim.x)

// NOTE: nvcc segfaults if these are included
// #include <CGAL/Delaunay_triangulation_2.h>
// #include <CGAL/Voronoi_diagram_2.h>

__device__ double square(double x) { return x * x; }
__device__ double determinant(double a00, double a01, double a10, double a11) {
  return a00 * a11 - a10 * a01;
}

__device__ Point2 circumcenter(double coord[3][2]) {
  double dqx = coord[1][0] - coord[0][0];
  double drx = coord[2][0] - coord[0][0];
  double dqy = coord[1][1] - coord[0][1];
  double dry = coord[2][1] - coord[0][1];

  double r2 = square(drx) + square(dry);
  double q2 = square(dqx) + square(dqy);
  double den = 2 * determinant(dqx, dqy, drx, dry);

  double dcx = determinant(dry, dqy, r2, q2) / den;
  double dcy = -determinant(drx, dqx, r2, q2) / den;

  return Point2{{dcx + coord[0][0], dcy + coord[0][1]}};
}

__global__ void DT2VDVertices(KerPoint2Array points,
                              KerTriArray input,
                              Point2 *output) {
  GRID_STRIDE_LOOP(index, input._num) {
    const Tri tri = input._arr[index];
    double coord[3][2];
    for (int i = 0; i < 3; i++) {
      const Point2 point = points._arr[tri._v[i]];
      coord[i][0] = point._p[0];
      coord[i][1] = point._p[1];
    }
    output[index] = circumcenter(coord);
  }
}

void extractCrust(int s_range, const TriHVec &input, SegmentHVec &output) {
  for (auto it = input.begin(); it != input.end(); it++) {
    const Tri tri = *it;
    if (tri._v[0] < s_range && tri._v[1] < s_range)
      output.push_back(Segment{tri._v[0], tri._v[1]});
    if (tri._v[1] < s_range && tri._v[2] < s_range)
      output.push_back(Segment{tri._v[1], tri._v[2]});
    if (tri._v[2] < s_range && tri._v[0] < s_range)
      output.push_back(Segment{tri._v[2], tri._v[0]});
  }
}

void GpuCurve::compute(const GCurve2DInput &input, GCurve2DOutput *output) {

  // Let S be a finite set of points in the plane.
  // move input from CPU to GPU
  _s_points.copyFromHost(input.pointVec);

  // Let V be the vertices of the Voronoi diagram of S.
  // Compute DT
  // CPU input → GPU output
  GDel2DInputGPU dt1Input{GDel2DInput{}, _s_points};
  GDel2DOutputGPU dt1Output;
  _v_gDel.computeGPU(dt1Input, &dt1Output);

  // convert to VD: compute circumcenter of triangles in GPU
  _v_points.resize(dt1Output.triVec.size());
  DT2VDVertices<<<1, 1>>>(toKernelArray(_s_points),
                          toKernelArray(dt1Output.triVec),
                          toKernelPtr(_v_points));
  CudaCheckError();

  // Let D be the Delaunay triangulation of S∪V.
  _sv_points.copyFrom2(_s_points, _v_points);
  GDel2DInputGPU dt2Input{GDel2DInput{}, _sv_points};
  GDel2DOutputGPU dt2Output;
  _sv_gDel.computeGPU(dt2Input, &dt2Output);

  // An edge of D belongs to the crust of S if both its endpoints belong to S
  // movo to cpu and extract crust
  TriHVec suv_tris;
  // dt2Output.triVec.copyToHost(suv_tris);
  extractCrust(_s_points.size(), suv_tris, output->segmentVec);
}
